#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <cmath>
#include<array>
#include <>
#include "Headers/deviceQuery.hpp"

////////////////////////////////////////////////////////////////////////////////
// Constants
#define PI 3.1415926536f
#define gameW 1200
#define gameH 900

#ifndef _OPERATIONS_CU_
#define _OPERATIONS_CU_

__global__ void asteroidPositionKernel(float* theta, float factor, float* outX, float* outY, int numElems) {
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (x< numElems) {
        float asteroidAngle = (PI / 180) * (theta[x]); // Convert asteroid's angle to radians
        float newX = std::cos(asteroidAngle) * factor;
        float newY = std::sin(asteroidAngle) * factor;

        outX[x] = newX;
        outY[x] = newY;
    }
}

__global__ void asteroidBoundsKernel(float* posX, float* posY, float* w, float* h, float* outPosX, float* outPosY, int numElems) {
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (x < numElems) {
        outPosX[x] = 99999;
        outPosY[x] = 99999;

        // Check X positions
        if (posX[x] + w[x] < 0)
        {
            outPosX[x] = gameW;
            outPosY[x] = posY[x];
        }
        if (posX[x] - w[x] > gameW)
        {
            outPosX[x] = -w[x];
            outPosY[x] = posY[x];
        }

        // Check Y positions
        if (posY[x] + h[x] < 0)
        {
            outPosX[x] = posX[x];
            outPosY[x] = gameH + h[x];
        }
        if (posY[x] - h[x] > gameH)
        {
            outPosX[x] = posX[x];
            outPosY[x] = -h[x];
        }
    }
}


__global__ void asteroidCheckBulletKernel(float* posX, float* posY, float* w, float* h, float* bull_l, float* bull_r, float* bull_t, float* bull_b, int* out, int numElems) {
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    out[x] = 99999;

    if (x < numElems) {
        for (int i = 0; i < sizeof(bull_l); i++) {
            if (bull_r[i] >= posX[x] && bull_l[i] <= (posX[x] + w[x]) && bull_t[i] >= posY[x] && bull_b[i] <= (posY[x] + h[x]))
            {
                out[x] = i;
            }
        }
    }
}


extern "C" void cuda_asteroidPos(float* theta, float factor, float* outX, float* outY, int numElems) {
    hipError_t error = hipSuccess;
    
    std::array<int, 9> cudaInfo = getCudaInfo();

    int blocksPerGrid = (numElems + cudaInfo[3] - 1) / cudaInfo[3];

    asteroidPositionKernel<<< blocksPerGrid, numElems >>> (theta, factor, outX, outY, numElems);

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("\nasteroidPositionKernel() failed to launch error = %d\n", error);
    }
}

extern "C" void cuda_asteroidBounds(float* posX, float* posY, float* w, float* h, float* outPosX, float* outPosY, int numElems) {
    hipError_t error = hipSuccess;

    std::array<int, 9> cudaInfo = getCudaInfo();
    int blocksPerGrid = (numElems + cudaInfo[3] - 1) / cudaInfo[3];

    asteroidBoundsKernel <<< blocksPerGrid, numElems >>> (posX, posY, w, h, outPosX, outPosY, numElems);

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("\asteroidBoundsKernel() failed to launch error = %d\n", error);
    }
}

extern "C" void cuda_asteroidCheckBullet(float* posX, float* posY, float* w, float* h, float* bull_l, float* bull_r, float* bull_t, float* bull_b, int* out, int numElems) {
    hipError_t error = hipSuccess;

    std::array<int, 9> cudaInfo = getCudaInfo();
    int blocksPerGrid = (numElems + cudaInfo[3] - 1) / cudaInfo[3];

    asteroidCheckBulletKernel <<< blocksPerGrid, numElems >>> (posX, posY, w, h, bull_l, bull_r, bull_t, bull_b, out, numElems);

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("\asteroidBoundsKernel() failed to launch error = %d\n", error);
    }
}

#endif